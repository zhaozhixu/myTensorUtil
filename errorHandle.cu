#include "errorHandle.h"

void checkError(hipError_t error)
{
     if (error == hipSuccess)
          return;
     fprintf(stderr, "CUDA_ERROR(%d) %s: %s\n", error, hipGetErrorName(error), hipGetErrorString(error));
     abort();
}
