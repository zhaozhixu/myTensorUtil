#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <time.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
/* #include "tensorCuda.h" */
#include "tensorUtil.h"
#include "errorHandle.h"
#include "sdt_alloc.h"

#define MAXDIM 8
#define MAX_THREADS_PER_BLOCK 1024
#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

/* static float EPSILON = 1e-16; */

static void assertTensor(const Tensor *tensor)
{
     assert(tensor && tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0);
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isTensorValid(const Tensor *tensor)
{
     return (tensor && tensor->data &&
             tensor->ndim < MAXDIM && tensor->ndim > 0 &&
             tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isShapeEqual(const Tensor *t1, const Tensor *t2)
{
     assertTensor(t1);
     assertTensor(t2);
     if (t1->ndim == t2->ndim) {
          int ndim = t1->ndim;
          while (--ndim >= 0)
               if (t1->dims[ndim] != t2->dims[ndim])
                    return 0;
          return 1;
     }
     return 0;
}

/* can only identify host memory alloced by hipHostMalloc, etc */
int isHostMem(const void *ptr)
{
     hipPointerAttribute_t attributes;
     checkError(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeHost;
}

int isDeviceMem(const void *ptr)
{
     hipPointerAttribute_t attributes;
     checkError(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeDevice;
}

void *cloneMem(const void *src, size_t size, CloneKind kind)
{
     assert(src);
     void *p;
     switch (kind) {
     case H2H:
          p = sdt_alloc(size);
          memmove(p, src, size);
          return p;
     case H2D:
          checkError(hipMalloc(&p, size));
          checkError(hipMemcpy(p, src, size, hipMemcpyHostToDevice));
          return p;
     case D2D:
          assert(isDeviceMem(src));
          checkError(hipMalloc(&p, size));
          checkError(hipMemcpy(p, src, size, hipMemcpyDeviceToDevice));
          return p;
     case D2H:
          assert(isDeviceMem(src));
          p = sdt_alloc(size);
          checkError(hipMemcpy(p, src, size, hipMemcpyDeviceToHost));
          return p;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }

}

Tensor *cloneTensor(const Tensor *src, CloneKind kind)
{
     assert(isTensorValid(src));
     float *data = (float *)cloneMem(src->data, src->len * sizeof(float), kind);
     Tensor *dst = createTensor(data, src->ndim, src->dims);
     return dst;
}

void *repeatMem(void *data, size_t size, int times, CloneKind kind)
{
     assert(data && times > 0);
     void *p, *dst;
     int i;
     switch (kind) {
     case H2H:
          dst = p = sdt_alloc(size * times);
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               memmove(p, data, size);
          return dst;
     case H2D:
          checkError(hipMalloc(&p, size * times));
          dst = p;
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyHostToDevice));
          return dst;
     case D2D:
          assert(isDeviceMem(data));
          checkError(hipMalloc(&p, size * times));
          dst = p;
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyDeviceToDevice));
          return dst;
     case D2H:
          assert(isDeviceMem(data));
          dst = p = sdt_alloc(size * times);
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyDeviceToHost));
          return dst;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }
}


int computeLength(int ndim, const int *dims)
{
     if (dims) {
          int i, len = 1;
          for (i = 0; i < ndim; i++)
               len *= dims[i];
          return len;
     }
     fprintf(stderr, "Warning: null dims in computeLength\n");
     return 0;
}

Tensor *createTensor(float *data, int ndim, const int *dims)
{
     Tensor *t = (Tensor *)sdt_alloc(sizeof(Tensor));
     t->data = data;
     t->ndim = ndim;
     t->dims = (int *)sdt_alloc(sizeof(int) * ndim);
     memmove(t->dims, dims, sizeof(int) * ndim);
     t->len = computeLength(ndim, dims);
     return t;
}

Tensor *mallocTensor(int ndim, const int* dims, const MallocKind mkind)
{
     Tensor *t = createTensor(NULL, ndim, dims);
     float *f;

     switch (mkind) {
     case HOST:
          f = (float *)sdt_alloc(t->len * sizeof(float));
          break;
     case DEVICE:
          checkError(hipMalloc(&f, t->len * sizeof(float)));
          break;
     default:
          fprintf(stderr, "unknown MallocKind %d\n", mkind);
          return NULL;
     }

     t->data = f;
     return t;
}

void freeTensor(Tensor *t, int do_free_data)
{
     assert(isTensorValid(t));
     sdt_free(t->dims);
     if (do_free_data) {
          if (isDeviceMem(t->data))
               checkError(hipFree(t->data));
          else
               sdt_free(t->data);
     }
     sdt_free(t);
}

void fprintTensor(FILE *stream, const Tensor *tensor, const char *fmt)
{
     assertTensor(tensor);
     int dim_sizes[MAXDIM], dim_levels[MAXDIM]; /* dimision size and how deep current chars go */
     int ndim = tensor->ndim, len = tensor->len, *dims = tensor->dims; /* pointer short cut */
     float *data = tensor->data;
     char left_buf[MAXDIM+1], right_buf[MAXDIM+1]; /* buffer for brackets */
     char *lp = left_buf, *rp = right_buf;
     size_t right_len;
     int i, j, k;

     dim_sizes[ndim-1] = tensor->dims[ndim-1];
     dim_levels[ndim-1] = 0;
     for (i = ndim-2; i >= 0; i--) {
          dim_sizes[i] = dims[i] * dim_sizes[i+1];
          dim_levels[i] = 0;
     }
     for (i = 0; i < len; i++) {
          for (j = 0; j < ndim; j++) {
               if (i % dim_sizes[j] == 0)
                    dim_levels[j]++;
               if (dim_levels[j] == 1) {
                    *lp++ = '[';
                    dim_levels[j]++;
               }
               if (dim_levels[j] == 3) {
                    *rp++ = ']';
                    if (j != 0 && dim_levels[j] > dim_levels[j-1]) {
                         *lp++ = '[';
                         dim_levels[j] = 2;
                    } else
                         dim_levels[j] = 0;
               }
          }
          *lp = *rp = '\0';
          fprintf(stream, "%s", right_buf);
          if (*right_buf != '\0') {
               fprintf(stream, "\n");
               right_len = strlen(right_buf);
               for (k = ndim-right_len; k > 0; k--)
                    fprintf(stream, " ");
          }
          fprintf(stream, "%s", left_buf);
          if (*left_buf == '\0')
               fprintf(stream, " ");
          fprintf(stream, fmt, data[i]);
          lp = left_buf, rp = right_buf;
     }
     for (j = 0; j < ndim; j++)
          fprintf(stream, "]");
     fprintf(stream, "\n");
}

void printTensor(const Tensor *tensor, const char *fmt)
{
     fprintTensor(stdout, tensor, fmt);
}

void fprintDeviceTensor(FILE *stream, const Tensor *d_tensor, const char *fmt)
{
     assert(isTensorValid(d_tensor));
     Tensor *h_tensor = cloneTensor(d_tensor, D2H);
     fprintTensor(stream, h_tensor, fmt);
     free(h_tensor->data); /* TODO: free t_tensor */
}

void printDeviceTensor(const Tensor *d_tensor, const char *fmt)
{
     fprintDeviceTensor(stdout, d_tensor, fmt);
}

void saveTensor(const char *file_name, const Tensor *tensor, const char *fmt)
{
     FILE *fp = fopen(file_name, "w");
     fprintTensor(fp, tensor, fmt);
     fclose(fp);
}

double getUnixTime(void)
{
     struct timespec tv;

     if(clock_gettime(CLOCK_REALTIME, &tv) != 0) return 0;

     return (tv.tv_sec + (tv.tv_nsec / 1.0e9));
}

Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len)
{
     assert(isTensorValid(src));
     assert(dim <= MAXDIM);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     checkError(hipMalloc(&dst->data, sizeof(float) * dst->len));
     return dst;
}

__global__ void sliceTensorKernel(float *src_data, float *dst_data, ...)
{
}

Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len)
{
     /* Your code here. You can have a cuda kernel below. */

     /* sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, ...) */

     return dst;
}
